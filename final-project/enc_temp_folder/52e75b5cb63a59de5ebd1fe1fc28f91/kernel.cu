#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "RGB.h"
#include <iostream>
#include "math.h"
#include <stdio.h>

/**
* Helper function to calculate the greyscale value based on R, G, and B
*/
__device__ int greyscale(BYTE red, BYTE green, BYTE blue)
{
	int grey = 0.3 * red + 0.59 * green + 0 * 11 * blue; // calculate grey scale
	return min(grey, 255);
}

/**
* Kernel for executing on GPY
*/
__global__ void greyscaleKernel(RGB* d_pixels, int height, int width)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height

	if (y >= height || y >= width)
		return;

	int index = y * width + x;

	int grey = greyscale(d_pixels[index].red, d_pixels[index].green, d_pixels[index].blue); // calculate grey scale

	d_pixels[index].red = grey;
	d_pixels[index].green = grey;
	d_pixels[index].blue = grey;

}

/**
*	Helper function to calculate the number of blocks on an axis based on the total grid size and number of threads in that axis
*/
__host__ int calcBlockDim(int total, int num_threads)
{
	int r = total / num_threads;
	if (total % num_threads != 0) // add one to cover all the threads per block
		++r;
	return r;
}

/**
*	Host function for launching greyscale kernel
*/
__host__ void d_convert_greyscale(RGB* pixel, int height, int width)
{
	RGB* d_pixel;

	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	greyscaleKernel << <grid, block >> > (d_pixel, height, width);

	hipMemcpy(pixel, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost);
}

__global__ void plusBlurKernel(RGB* d_pixels, int height, int width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height
	int index = y * width + x;

	int sumRed = d_pixels[index].red;
	int sumBlue = d_pixels[index].blue;
	int sumGreen = d_pixels[index].green;
	int numOfChanges = 1;

	if (y >= height || y >= width)
		return;
	if (y + 1 < height) {
		int i = (y + 1) * width + x;
		sumRed += d_pixels[i].red;
		sumBlue += d_pixels[i].blue;
		sumGreen += d_pixels[i].green;
		numOfChanges++;
	}
	if (y - 1 > 0) {
		int i = (y - 1) * width + x;
		sumRed += d_pixels[i].red;
		sumBlue += d_pixels[i].blue;
		sumGreen += d_pixels[i].green;
		numOfChanges++;
	}
	if (x + 1 < width) {
		int i = y * width + (x + 1);
		sumRed += d_pixels[i].red;
		sumBlue += d_pixels[i].blue;
		sumGreen += d_pixels[i].green;
		numOfChanges++;
	} if (x - 1 > 0) {
		int i = y * width + (x - 1);
		sumRed += d_pixels[i].red;
		sumBlue += d_pixels[i].blue;
		sumGreen += d_pixels[i].green;
		numOfChanges++;
	}

	d_pixels[index].red = sumRed / numOfChanges;
	d_pixels[index].green = sumGreen / numOfChanges;
	d_pixels[index].blue = sumBlue / numOfChanges;
}

__host__ void plusBlurLauncher(RGB* pixel, int height, int width) {
	RGB* d_pixel;

	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	plusBlurKernel << <grid, block >> > (d_pixel, height, width);

	hipMemcpy(pixel, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost);

}

__global__ void  squareBlurKernel(RGB* pixels, int height, int width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height

	int index = y * width + x;

	int sumRed = pixels[index].red;
	int sumBlue = pixels[index].blue;
	int sumGreen = pixels[index].green;
	int numOfChanges = 1;

	if (y >= height || y >= width) {
		return;
	}

	//use a double for loop, and loop through the area adding into the averages while it is within bounds

	for (int i = -2; i < 2; i++) {
		for (int j = -2; j < 2; j++) {
			if ((x + i > 0 && y + j > 0) && (x + i < width && y + j < height)) {
				int is = (y + j) * width + (x + i);
				sumRed += pixels[is].red;
				sumBlue += pixels[is].blue;
				sumGreen += pixels[is].green;
				numOfChanges++;
			}

		}
	}
	pixels[index].red = sumRed / numOfChanges;
	pixels[index].green = sumGreen / numOfChanges;
	pixels[index].blue = sumBlue / numOfChanges;
}

//for edgeDir and gradiant I will have to use the index shit that has been used in all of the above stuff
__global__ void GradiantStrength(RGB* pixels, int* edgeDir, int* gradiant, int height, int width) {
	//printf("hi");
	int row = blockIdx.x * blockDim.x + threadIdx.x; // width
	int col = blockIdx.y * blockDim.y + threadIdx.y; // height
	int newAngle = 0;
	int index = col * width + row;
	int* GxMask = (int*)malloc(width*3);
	int* GyMask = (int*)malloc(width*3);
	
	//int GxMask[1000000];				// Sobel mask in the x direction
	//int GyMask[1000000];				// Sobel mask in the y direction
	//printf("hello there");
	//sobel mask set up
	GxMask[width * 4] = 20;

	GxMask[0] = -1; GxMask[1] = -2;  GxMask[2] = -1;
	GxMask[width] = 0;  GxMask[width + 1] = 0;  GxMask[width + 2] = 0;
	GxMask[width * 2] = 1;  GxMask[width * 2 + 1] = 2;  GxMask[width * 2 + 2] = 1;
	GyMask[0] = 1; GyMask[1] = 0; GyMask[2] = -1;
	GyMask[width] = 2; GyMask[width + 1] = 0; GyMask[width + 2] = -2;
	GyMask[width * 2] = 1; GyMask[width * 2 + 1] = 0; GyMask[width * 2 + 2] = -1;
	
	printf("ahhhhh");
	if (col >= height || col >= width) {
		return;
	}
	printf("a");
	//long i = (unsigned long)(row * 3 * width + 3 * col);
	double Gx = 0;
	double Gy = 0;
	/* Calculate the sum of the Sobel mask times the nine surrounding pixels in the x and y direction */
	for (int rowOffset = -1; rowOffset <= 1; rowOffset++) {
		for (int colOffset = -1; colOffset <= 1; colOffset++) {
			int rowTotal = row + rowOffset;
			int colTotal = col + colOffset;
			long iOffset = (unsigned long)(rowTotal * 3 * width + colTotal * 3);

			int gIndex = colOffset * width + rowOffset + 1;
			Gx = Gx + (pixels[index].red * (GxMask[gIndex]));//the image should have already been changed to grayscale so any color should be fine
			//std::cout << "red from graidnant strength in kernel is: " << pixels[index].red << std::endl;
			printf("red from graidnant strength in kernel is: %d", pixels[index].red);
			Gy = Gy + (pixels[index].red * (GyMask[gIndex]));
		}
	}

	gradiant[index] = sqrt((Gx*Gx) + (Gy*Gy));	// Calculate gradient strength			
	
	double thisAngle = (atan2(Gx, Gy) / 3.14159) * 180.0;		// Calculate actual direction of edge

	/* Convert actual edge direction to approximate value */
	if (((thisAngle < 22.5) && (thisAngle > -22.5)) || (thisAngle > 157.5) || (thisAngle < -157.5))
		newAngle = 0;
	if (((thisAngle > 22.5) && (thisAngle < 67.5)) || ((thisAngle < -112.5) && (thisAngle > -157.5)))
		newAngle = 45;
	if (((thisAngle > 67.5) && (thisAngle < 112.5)) || ((thisAngle < -67.5) && (thisAngle > -112.5)))
		newAngle = 90;
	if (((thisAngle > 112.5) && (thisAngle < 157.5)) || ((thisAngle < -22.5) && (thisAngle > -67.5)))
		newAngle = 135;

	edgeDir[index] = newAngle;		// Store the approximate edge direction of each pixel in one array
}

__host__ void squareBlurLauncher(RGB* pixel, int height, int width) {
	RGB* d_pixel;

	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	squareBlurKernel << <grid, block >> > (d_pixel, height, width);

	hipMemcpy(pixel, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost);
}

__host__ void gradiantLauncher(RGB* pixels, int* edgeDir, int* gradiant, int height, int width) {
	RGB* d_pixel;

	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixels, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	GradiantStrength << <grid, block >> > (d_pixel, edgeDir, gradiant, height, width);
	hipMemcpy(pixels, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost);
}

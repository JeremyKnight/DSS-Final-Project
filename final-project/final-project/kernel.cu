#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "RGB.h"
#include <iostream>

/**
* Helper function to calculate the greyscale value based on R, G, and B
*/
__device__ int greyscale(BYTE red, BYTE green, BYTE blue)
{
	int grey = 0.3 * red + 0.59 * green + 0 * 11 * blue; // calculate grey scale
	return min(grey, 255);
}

/**
* Kernel for executing on GPY
*/
__global__ void greyscaleKernel(RGB* d_pixels, int height, int width)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height

	if (y >= height || y >= width)
		return;

	int index = y * width + x;

	int grey = greyscale(d_pixels[index].red, d_pixels[index].green, d_pixels[index].blue); // calculate grey scale

	d_pixels[index].red = grey;
	d_pixels[index].green = grey;
	d_pixels[index].blue = grey;

}

/**
*	Helper function to calculate the number of blocks on an axis based on the total grid size and number of threads in that axis
*/
__host__ int calcBlockDim(int total, int num_threads)
{
	int r = total / num_threads;
	if (total % num_threads != 0) // add one to cover all the threads per block
		++r;
	return r;
}

/**
*	Host function for launching greyscale kernel
*/
__host__ void d_convert_greyscale(RGB* pixel, int height, int width)
{
	RGB* d_pixel;

	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	greyscaleKernel << <grid, block >> > (d_pixel, height, width);

	hipMemcpy(pixel, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost);
}

__global__ void plusBlurKernel(RGB* d_pixels, int height, int width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height

	int index = y * width + x;

	int sumRed = d_pixels[index].red;
	int sumBlue = d_pixels[index].blue;
	int sumGreen = d_pixels[index].green;
	int numOfChanges = 1;

	if (y >= height || y >= width)
		return;
	if (y + 1 < height) {
		int i = (y + 1) * width + x;
		sumRed += d_pixels[i].red;
		sumBlue += d_pixels[i].blue;
		sumGreen += d_pixels[i].green;
		numOfChanges++;
	}
	if (y - 1 > 0) {
		int i = (y - 1) * width + x;
		sumRed += d_pixels[i].red;
		sumBlue += d_pixels[i].blue;
		sumGreen += d_pixels[i].green;
		numOfChanges++;
	}
	if (x + 1 < width) {
		int i = y * width + (x + 1);
		sumRed += d_pixels[i].red;
		sumBlue += d_pixels[i].blue;
		sumGreen += d_pixels[i].green;
		numOfChanges++;
	} if (x - 1 > 0) {
		int i = y * width + (x - 1);
		sumRed += d_pixels[i].red;
		sumBlue += d_pixels[i].blue;
		sumGreen += d_pixels[i].green;
		numOfChanges++;
	}

	d_pixels[index].red = sumRed / numOfChanges;
	d_pixels[index].green = sumGreen / numOfChanges;
	d_pixels[index].blue = sumBlue / numOfChanges;
}

__host__ void plusBlurLauncher(RGB* pixel, int height, int width) {
	RGB* d_pixel;

	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	plusBlurKernel << <grid, block >> > (d_pixel, height, width);

	hipMemcpy(pixel, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost);

}

__global__ void  squareBlurKernel(RGB* pixels, int height, int width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height

	int index = y * width + x;

	int sumRed = pixels[index].red;
	int sumBlue = pixels[index].blue;
	int sumGreen = pixels[index].green;
	int numOfChanges = 1;

	if (y >= height || y >= width) {
		return;
	}

	//use a double for loop, and loop through the area adding into the averages while it is within bounds

	for (int i = -2; i < 2; i++) {
		for (int j = -2; j < 2; j++) {
			if ((x + i > 0 && y + j > 0) && (x + i < width && y + j < height)) {
				int is = (y + j) * width + (x + i);
				sumRed += pixels[is].red;
				sumBlue += pixels[is].blue;
				sumGreen += pixels[is].green;
				numOfChanges++;
			}

		}
	}
	pixels[index].red = sumRed / numOfChanges;
	pixels[index].green = sumGreen / numOfChanges;
	pixels[index].blue = sumBlue / numOfChanges;
}

__host__ void squareBlurLauncher(RGB* pixel, int height, int width) {
	RGB* d_pixel;

	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	squareBlurKernel << <grid, block >> > (d_pixel, height, width);

	hipMemcpy(pixel, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost);

}


// |Gx(x,y)| = -P(x-1,y-1) + -2 *P(x-1,y) + -P(x-1,y+1) + P(x+1,y-1) + 2 * P(x + 1, y) + P(x + 1, y + 1)
// |Gy(x,y)| = P(x-1,y-1) + 2*P(x,y-1) + P(x+1,y-1) + -P(x-1,y+1) + –2 * P(x, y + 1) - P(x + 1, y + 1)

__global__ void edgeDetectionKernel(RGB* d_pixels, int height, int width)
{
	// determine the current pixel
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	/*float h, s, i;
	i = (d_pixels[row * width + col].red + d_pixels[row * width + col].green + d_pixels[row * width + col].blue) / 3;
	h = acos(((d_pixels[row * width + col].red - d_pixels[row * width + col].green) + (d_pixels[row * width + col].red - d_pixels[row * width + col].blue) / (2 * sqrt(((d_pixels[row * width + col].red - d_pixels[row * width + col].green) * (d_pixels[row * width + col].red - d_pixels[row * width + col].green)) + ((d_pixels[row * width + col].red - d_pixels[row * width + col].blue) * (d_pixels[row * width + col].green - d_pixels[row * width + col].blue))))));
	if (d_pixels[row * width + col].red < d_pixels[row * width + col].green && d_pixels[row * width + col].red < d_pixels[row * width + col].blue) {
		s = d_pixels[row * width + col].red / i;
	}
	else if (d_pixels[row * width + col].green < d_pixels[row * width + col].red && d_pixels[row * width + col].green < d_pixels[row * width + col].blue) {
		s = d_pixels[row * width + col].green / i;
	}
	else {
		s = d_pixels[row * width + col].blue / i;
	}

	d_pixels[row * width + col].red = h;
	d_pixels[row * width + col].green = s;
	d_pixels[row * width + col].blue = i;*/

	float reddx, reddy;
	float greendx, greendy;
	float bluedx, bluedy;

	if (col < width && row < height) {
		if (col > 0 && row > 0 && col < width && row < height) {
			//red
			reddx = (-1 * d_pixels[(row - 1) * width + (col - 1)].red) + (-2 * d_pixels[row * width + (col - 1)].red) + (-1 * d_pixels[(row + 1) * width + (col - 1)].red) + (d_pixels[(row - 1) * width + (col + 1)].red) + (2 * d_pixels[row * width + (col + 1)].red) + (d_pixels[(row + 1) * width + (col + 1)].red);
			reddy = (d_pixels[(row - 1) * width + (col - 1)].red) + (2 * d_pixels[(row - 1) * width + col].red) + (d_pixels[(row - 1) * width + (col + 1)].red) + (-1 * d_pixels[(row + 1) * width + (col - 1)].red) + (-2 * d_pixels[(row + 1) * width + col].red) + (-1 * d_pixels[(row + 1) * width + (col + 1)].red);
			
			//reddx /= 5;
			//reddy /= 5;
			
			d_pixels[row * width + col].red = sqrt((reddx * reddx) + (reddy * reddy));
			
			//green
			greendx = (-1 * d_pixels[(row - 1) * width + (col - 1)].green) + (-2 * d_pixels[row * width + (col - 1)].green) + (-1 * d_pixels[(row + 1) * width + (col - 1)].green) + (d_pixels[(row - 1) * width + (col + 1)].green) + (2 * d_pixels[row * width + (col + 1)].green) + (d_pixels[(row + 1) * width + (col + 1)].green);
			greendy = (d_pixels[(row - 1) * width + (col - 1)].green) + (2 * d_pixels[(row - 1) * width + col].green) + (d_pixels[(row - 1) * width + (col + 1)].green) + (-1 * d_pixels[(row + 1) * width + (col - 1)].green) + (-2 * d_pixels[(row + 1) * width + col].green) + (-1 * d_pixels[(row + 1) * width + (col + 1)].green);
			
			//greendx /= 5;
			//greendy /= 5;
			
			d_pixels[row * width + col].green = sqrt((greendx * greendx) + (greendy * greendy));
			
			//blue
			bluedx = (-1 * d_pixels[(row - 1) * width + (col - 1)].blue) + (-2 * d_pixels[row * width + (col - 1)].blue) + (-1 * d_pixels[(row + 1) * width + (col - 1)].blue) + (d_pixels[(row - 1) * width + (col + 1)].blue) + (2 * d_pixels[row * width + (col + 1)].blue) + (d_pixels[(row + 1) * width + (col + 1)].blue);
			bluedy = (d_pixels[(row - 1) * width + (col - 1)].blue) + (2 * d_pixels[(row - 1) * width + col].blue) + (d_pixels[(row - 1) * width + (col + 1)].blue) + (-1 * d_pixels[(row + 1) * width + (col - 1)].blue) + (-2 * d_pixels[(row + 1) * width + col].blue) + (-1 * d_pixels[(row + 1) * width + (col + 1)].blue);
			
			//bluedx /= 5;
			//bluedy /= 5;
			
			d_pixels[row * width + col].blue = sqrt((bluedx * bluedx) + (bluedy * bluedy));
		}
	}
}

__host__ void d_edge_detection(RGB* pixel, int height, int width)
{
	RGB* d_pixel;

	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	edgeDetectionKernel << <grid, block >> > (d_pixel, height, width);

	hipMemcpy(pixel, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost);
}
